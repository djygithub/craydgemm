#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
/*
CUDA Tutorial, matrix-matrix multiply
UC Berkeley Reactor Design and Neutronics Group
Ryan M. Bergmann - 1/22/2014
*/
#define BILLION  1E9

#define CHECK(cmd) \
{\
    hipError_t error  = cmd;\
    if (error != hipSuccess) { \
        fprintf(stderr, "error: '%s'(%d) at %s:%d\n", hipGetErrorString(error), error,__FILE__, __LINE__); \
        exit(EXIT_FAILURE);\
          }\
}



void matmul_cpu(long len, double* a , double* b , double* c){
/*
	// initialize local variable to hold values while the sum is done
	double sum;
	unsigned row,col,k;

	for(col=0 ; col<len ; col++ ){       //scan the rows
		for(row=0 ; row<len ; row++ ){   //scan the cols

			// zero out sum
			sum = 0;

			// scan the row of a, the col of b
			for(k=0;k<len;k++){
				sum +=  a[ row * len + k ] * b[ k * len + col ];
			}

			// write final value into output array
			c[ len * row + col  ] = sum;

		}
	}

*/
}

__global__ void matmul_kernel( long len, double* a , double* b , double* c){

	//
	//  THIS IS THE SIMPLE WAY TO DO IT, NOT THE ***FAST WAY*** -> uses 2*N^3 global loads
	//

	// get index in c
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	//return if over the length
	if(row>=len | col>=len){return;}

	// initialize local variable to hold values while the sum is done
	double sum = 0;
	unsigned j;

	// scan the row of a, the col of b
	for(j=0;j<len;j++){
		sum +=  a[ row * len + j ] * b[ j * len + col ];
	}

	// write final value into output array
	c[ len * row + col  ] = sum;

}


int get_time(){

	return ((int)clock())/((int)CLOCKS_PER_SEC);

}

int main(int argc, char *argv[]){

	// declare
	double* 		a;
	double*  	b;  
	double* 		c;
	double* 		d_a;
	double*		d_b;
	double*		d_c;
	long            N = atol(argv[1]);
	long  	        len_a=N, len_b=N, j, k;
	long 		bytes_a, bytes_b, bytes_c, shared_mem_size;
	dim3 		NUM_THREADS, blks;
	double 		accum;
  double numiops = (N * N * N) * 2;
  double cpu_elapsed;
  double gpu_elapsed;
  double cpugiops;
  double gpugiops;
	hipEvent_t 	start, stop;
	float		time;
	struct timeval  tvalBefore, tvalAfter;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	printf("------ Matrix Dimensions ------\n");
	printf("dims a,b = %ld , %ld \n",len_a,len_b);
	assert(len_a==len_b);
	bytes_a = len_a * len_a * sizeof(double);
	bytes_b = len_b * len_b * sizeof(double);
	bytes_c = len_b * len_b * sizeof(double);
        
        printf ("info: allocate host mem (%6.2f MB)\n", (3.0*bytes_a)/1024.0/1024.0);
	//allocate arrays
	//a = (int*) malloc( bytes_a );
	hipHostAlloc(&a,bytes_a,hipHostMallocDefault);
	CHECK(a == 0 ? hipErrorOutOfMemory : hipSuccess );
	//b = (int*) malloc( bytes_b );
	hipHostAlloc(&b,bytes_b,hipHostMallocDefault);
	CHECK(b == 0 ? hipErrorOutOfMemory : hipSuccess );
	//c = (int*) malloc( bytes_b );
	hipHostAlloc(&c,bytes_c,hipHostMallocDefault);
	CHECK(c == 0 ? hipErrorOutOfMemory : hipSuccess );
        printf ("info: device  mem (%6.2f MB)\n", (3.0*bytes_a)/1024.0/1024.0);
	//allocate device arrays
	CHECK(hipMalloc( &d_a , bytes_a ));  //must be pointer to the point, since the actual point value is being changed, not the value it points to
	CHECK(hipMalloc( &d_b , bytes_b ));
	CHECK(hipMalloc( &d_c , bytes_c ));
         
	if(a==NULL || b == NULL || c == NULL )
	printf("Could not allocate host memory\n");
        printf("Filling in 2D arrays a and b \n");
	// read in data
	for(j=0;j<len_a;j++){
		for(k=0;k<len_a;k++){
			a[j*len_a+k]= 2;  //row major
			b[j*len_a+k] = 2;
		}
	}

        printf("Filling Complete\n");
	// determine gpu parameters, print them
	NUM_THREADS.x   = NUM_THREADS.y = 16;
	blks.x = blks.y = (len_a + NUM_THREADS.x - 1 ) / NUM_THREADS.x;
	NUM_THREADS.z   = blks.z = 1;
	printf("------- CUDA Parameters -------\n");
	printf("NUM_THREADS(%4u,%4u,   0)\n       blks(%4u,%4u,   0)\n",NUM_THREADS.x,NUM_THREADS.y,blks.x,blks.y);
  printf("TOTAL DBLOPS %lf \n",numiops);
	printf("-------------------------------\n");

	//launch cpu version to compare
        printf("\nCalling CPU Matrix Multiply \n");
	gettimeofday (&tvalBefore, NULL);
	matmul_cpu(len_a, a, b, c);
	gettimeofday (&tvalAfter, NULL);
	printf("\nCPU took %lf seconds as computed by gettimeofday() function\n",
           ( ((tvalAfter.tv_sec - tvalBefore.tv_sec)*1000000L
           +tvalAfter.tv_usec) - tvalBefore.tv_usec
          )/1000000.00);

  
  

/*  cpu_elapsed = ( ((tvalAfter.tv_sec - tvalBefore.tv_sec)*1000000L
           +tvalAfter.tv_usec) - tvalBefore.tv_usec
          )/1000000.00; */
  /* printf("\nCPU ELAPSED %lf \n",cpu_elapsed); */
/*  cpugiops = (numiops / cpu_elapsed)/1000000000;
  printf("CPUGIOPS %lf \n",cpugiops); */

        printf("\nCPU Matrix multiplication completed. Time to launch GPU kernel.\n");
	hipEventRecord(start, 0);

        hipMemcpy( d_a , a , bytes_a , hipMemcpyHostToDevice );
        hipMemcpy( d_b , b , bytes_b , hipMemcpyHostToDevice );


        matmul_kernel<<< blks, NUM_THREADS>>> (len_a , d_a , d_b , d_c);
        hipMemcpy( b , d_c , bytes_b , hipMemcpyDeviceToHost );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("\nGPU took %f seconds as computed by CudaEvent function\n", time/1000);
  gpugiops = (numiops / time)/1000000;
  printf ("GPU-GDBLOPS/second %lf \n",gpugiops);

	if(hipPeekAtLastError()){
		printf("CUDA ERROR, %s\n",hipGetErrorString(hipPeekAtLastError()));
		return 1;
	}


	printf("\nExperiment Done.\n");
	printf("-------------------------------\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	// return zero if all ok
	return 0;

}
