#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h> 
#include <algorithm>
#include <iostream>
#include <windows.h>
using namespace std;

/* #include <sys/time.h> */

/*
CUDA Tutorial, matrix-matrix multiply
UC Berkeley Reactor Design and Neutronics Group
Ryan M. Bergmann - 1/22/2014
C:\dellmatmul\double>nvcc -o MatMulDblGpuWin.exe MatMulDblGpuWin.cu
MatMulDblGpuWin.cu
   Creating library MatMulDblGpuWin.lib and object MatMulDblGpuWin.exp
C:\dellmatmul\double>MatMulDblGpuWin.exe 2000
------ Matrix Dimensions ------
dims a,b = 2000 , 2000
info: allocate host mem ( 91.55 MB)
info: device  mem ( 91.55 MB)
Filling in 2D arrays a and b
Filling Complete
------- CUDA Parameters -------
NUM_THREADS(  16,  16,   0)
       blks( 125, 125,   0)
TOTAL GFLOPS 16.000000
-------------------------------
CPU took 72.563000 seconds as computed by gettickcount
CPU-DOUBLE-GFLOPS/second 0.220498

CPU Matrix multiplication completed. Time to launch GPU kernel.

GPU took 0.632657 seconds as computed by CudaEvent function
GPU-DOUBLE-GFLOPS/second 25.290170

Experiment Done.
-------------------------------

C:\dellmatmul\double>

*/
#define BILLION  1E9

#define CHECK(cmd) \
{\
    hipError_t error  = cmd;\
    if (error != hipSuccess) { \
        fprintf(stderr, "error: '%s'(%d) at %s:%d\n", hipGetErrorString(error), error,__FILE__, __LINE__); \
        exit(EXIT_FAILURE);\
          }\
}



void matmul_cpu(long len, double* a , double* b , double* c){

	// initialize local variable to hold values while the sum is done
	double sum;
	unsigned row,col,k;

	for(col=0 ; col<len ; col++ ){       //scan the rows
		for(row=0 ; row<len ; row++ ){   //scan the cols

			// zero out sum
			sum = 0;

			// scan the row of a, the col of b
			for(k=0;k<len;k++){
				sum +=  a[ row * len + k ] * b[ k * len + col ];
			}

			// write final value into output array
			c[ len * row + col  ] = sum;

		}
	}


}

__global__ void matmul_kernel( long len, double* a , double* b , double* c){

	//
	//  THIS IS THE SIMPLE WAY TO DO IT, NOT THE ***FAST WAY*** -> uses 2*N^3 global loads
	//

	// get index in c
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	//return if over the length
	if(row>=len | col>=len){return;}

	// initialize local variable to hold values while the sum is done
	double sum = 0;
	unsigned j;

	// scan the row of a, the col of b
	for(j=0;j<len;j++){
		sum +=  a[ row * len + j ] * b[ j * len + col ];
	}

	// write final value into output array
	c[ len * row + col  ] = sum;

}


int get_time(){

	return ((int)clock())/((int)CLOCKS_PER_SEC);

}

int main(int argc, char *argv[]){

	// declare
	double* 		a;
	double*  	b;  
	double* 		c;
	double* 		d_a;
	double*		d_b;
	double*		d_c;
	long            N = atol(argv[1]);
        double          matrixsize = N;
	long  	        len_a=N, len_b=N, j, k;
	long 		bytes_a, bytes_b, bytes_c;
	dim3 		NUM_THREADS, blks;
        double gflops = ((matrixsize /1000) * (matrixsize / 1000) * (matrixsize / 1000) * 2);
        double cpugiops;
        double gpugiops;
	hipEvent_t 	start, startcpu, stop, stopcpu;
	float		time;
  
	hipEventCreate(&start);
	hipEventCreate(&stop);
        hipEventCreate(&startcpu);
	hipEventCreate(&stopcpu);


	printf("------ Matrix Dimensions ------\n");
	printf("dims a,b = %ld , %ld \n",len_a,len_b);
	assert(len_a==len_b);
	bytes_a = len_a * len_a * sizeof(double);
	bytes_b = len_b * len_b * sizeof(double);
	bytes_c = len_b * len_b * sizeof(double);
        
        printf ("info: allocate host mem (%6.2f MB)\n", (3.0*bytes_a)/1024.0/1024.0);
	//allocate arrays
	//a = (int*) malloc( bytes_a );
	hipHostAlloc(&a,bytes_a,hipHostMallocDefault);
	CHECK(a == 0 ? hipErrorOutOfMemory : hipSuccess );
	//b = (int*) malloc( bytes_b );
	hipHostAlloc(&b,bytes_b,hipHostMallocDefault);
	CHECK(b == 0 ? hipErrorOutOfMemory : hipSuccess );
	//c = (int*) malloc( bytes_b );
	hipHostAlloc(&c,bytes_c,hipHostMallocDefault);
	CHECK(c == 0 ? hipErrorOutOfMemory : hipSuccess );
        printf ("info: device  mem (%6.2f MB)\n", (3.0*bytes_a)/1024.0/1024.0);
	//allocate device arrays
	CHECK(hipMalloc( &d_a , bytes_a ));  //must be pointer to the point, since the actual point value is being changed, not the value it points to
	CHECK(hipMalloc( &d_b , bytes_b ));
	CHECK(hipMalloc( &d_c , bytes_c ));
         
	if(a==NULL || b == NULL || c == NULL )
	printf("Could not allocate host memory\n");
        printf("Filling in 2D arrays a and b \n");
	// read in data
	for(j=0;j<len_a;j++){
		for(k=0;k<len_a;k++){
			a[j*len_a+k]= 2;  //row major
			b[j*len_a+k] = 2;
		}
	}

        printf("Filling Complete\n");
	// determine gpu parameters, print them
	NUM_THREADS.x   = NUM_THREADS.y = 16;
	blks.x = blks.y = (len_a + NUM_THREADS.x - 1 ) / NUM_THREADS.x;
	NUM_THREADS.z   = blks.z = 1;
	printf("------- CUDA Parameters -------\n");
	printf("NUM_THREADS(%4u,%4u,   0)\n       blks(%4u,%4u,   0)\n",NUM_THREADS.x,NUM_THREADS.y,blks.x,blks.y);
        printf("TOTAL GFLOPS %lf \n",gflops);
	printf("-------------------------------\n");

	//launch cpu version to compare
  
	 /* gettimeofday (&tvalBefore, NULL); */
   	
        DWORD dw1 = GetTickCount();
	matmul_cpu(len_a, a, b, c);
        DWORD dw2 = GetTickCount();
        double result = dw2-dw1;
        printf("CPU took %f seconds as computed by gettickcount\n", result/1000);
        cpugiops = gflops / (result/1000);
        printf ("CPU-DOUBLE-GFLOPS/second %lf \n",cpugiops); 
        printf("\nCPU Matrix multiplication completed. Time to launch GPU kernel.\n");
	hipEventRecord(start, 0);

        hipMemcpy( d_a , a , bytes_a , hipMemcpyHostToDevice );
        hipMemcpy( d_b , b , bytes_b , hipMemcpyHostToDevice );


        matmul_kernel<<< blks, NUM_THREADS>>> (len_a , d_a , d_b , d_c);
        hipMemcpy( b , d_c , bytes_b , hipMemcpyDeviceToHost );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("\nGPU took %f seconds as computed by CudaEvent function\n", time/1000);
        gpugiops = gflops / (time/1000);
        printf ("GPU-DOUBLE-GFLOPS/second %lf \n",gpugiops);

	if(hipPeekAtLastError()){
		printf("CUDA ERROR, %s\n",hipGetErrorString(hipPeekAtLastError()));
		return 1;
	}


	printf("\nExperiment Done.\n");
	printf("-------------------------------\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	// return zero if all ok
	return 0;

}
